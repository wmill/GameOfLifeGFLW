#include "hip/hip_runtime.h"
#include <iostream>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <random>
#include <thread>
#include <string.h>

const int SCREEN_WIDTH = 1500;
const int SCREEN_HEIGHT = 1000;
const int IMAGE_WIDTH = SCREEN_WIDTH;
const int IMAGE_HEIGHT = SCREEN_HEIGHT;
const int NUM_THREADS = 16;
const int USE_GOOD_RANDOM = true;

typedef uint32_t Pixel;

const Pixel LIVE = 0xFF000000;
const Pixel DEAD = 0xFFFFFFFF;

__device__ Pixel getCell(Pixel* imageData, int h, int w) {
    return imageData[(h + IMAGE_HEIGHT) % IMAGE_HEIGHT * IMAGE_WIDTH + (w + IMAGE_WIDTH) % IMAGE_WIDTH];
}

__device__ int countNeighbors(Pixel* imageData, int h, int w) {
    int count = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) continue;
            count += getCell(imageData, h + i, w + j) == LIVE ? 1 : 0;
        }
    }
    return count;
}

__global__ void updateImageKernel(Pixel* greenImageData, Pixel* redImageData) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= IMAGE_WIDTH * IMAGE_HEIGHT) return;

    int h = idx / IMAGE_WIDTH;
    int w = idx % IMAGE_WIDTH;
    int neighbors = countNeighbors(greenImageData, h, w);
    Pixel cell = getCell(greenImageData, h, w);
    
    if (cell == LIVE) {
        if (neighbors < 2 || neighbors > 3) {
            redImageData[h * IMAGE_WIDTH + w] = DEAD;
        } else {
            redImageData[h * IMAGE_WIDTH + w] = LIVE;
        }
    } else {
        if (neighbors == 3) {
            redImageData[h * IMAGE_WIDTH + w] = LIVE;
        } else {
            redImageData[h * IMAGE_WIDTH + w] = DEAD;
        }
    }
}

void randomizeImage(Pixel* imageData) {
    static std::random_device rd;
    static std::mt19937 eng(rd());
    static std::uniform_int_distribution<> distr(0, 4);
    for (int y = 0; y < IMAGE_HEIGHT; ++y) {
        for (int x = 0; x < IMAGE_WIDTH; ++x) {
            int pixelIndex = y * IMAGE_WIDTH + x;
            imageData[pixelIndex] = distr(eng) == 0 ? LIVE : DEAD;
        }
    }
}

int main() {
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW\n";
        return -1;
    }

    GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "OpenGL Image Processing", NULL, NULL);
    if (!window) {
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    GLuint textureID;
    glGenTextures(1, &textureID);
    glBindTexture(GL_TEXTURE_2D, textureID);

    Pixel* imageDataA = new Pixel[IMAGE_WIDTH * IMAGE_HEIGHT];
    Pixel* imageDataB = new Pixel[IMAGE_WIDTH * IMAGE_HEIGHT];
    memset(imageDataA, LIVE, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel));
    memset(imageDataB, DEAD, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel));

    randomizeImage(imageDataA);

    Pixel *d_imageDataA, *d_imageDataB;
    hipMalloc(&d_imageDataA, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel));
    hipMalloc(&d_imageDataB, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel));
    hipMemcpy(d_imageDataA, imageDataA, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel), hipMemcpyHostToDevice);
    hipMemcpy(d_imageDataB, imageDataB, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel), hipMemcpyHostToDevice);

    hipGraphicsResource *cudaResource;
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, IMAGE_WIDTH, IMAGE_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, imageDataA);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

    while (!glfwWindowShouldClose(window)) {
        dim3 threadsPerBlock(256);
        dim3 blocksPerGrid((IMAGE_WIDTH * IMAGE_HEIGHT + threadsPerBlock.x - 1) / threadsPerBlock.x);

        hipGraphicsMapResources(1, &cudaResource, 0);
        hipArray_t hipArray;
        hipGraphicsSubResourceGetMappedArray(&hipArray, cudaResource, 0, 0);
        hipMemcpyToArray(hipArray, 0, 0, d_imageDataA, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(Pixel), hipMemcpyDeviceToDevice);
        hipGraphicsUnmapResources(1, &cudaResource, 0);

        updateImageKernel<<<blocksPerGrid, threadsPerBlock>>>(d_imageDataA, d_imageDataB);
        hipDeviceSynchronize();
        std::swap(d_imageDataA, d_imageDataB);

        glClear(GL_COLOR_BUFFER_BIT);
        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, textureID);
        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
        glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, -1.0f);
        glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
        glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, 1.0f);
        glEnd();
        glDisable(GL_TEXTURE_2D);

        glfwSwapBuffers(window);
        glfwPollEvents();

        GLenum err;
        while ((err = glGetError()) != GL_NO_ERROR) {
            std::cerr << "OpenGL error: " << err << std::endl;
        }
    }

    hipGraphicsUnregisterResource(cudaResource);
    hipFree(d_imageDataA);
    hipFree(d_imageDataB);
    delete[] imageDataA;
    delete[] imageDataB;

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
